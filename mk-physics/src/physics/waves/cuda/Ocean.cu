#include "hip/hip_runtime.h"
#include "Ocean_cu.h"

#include <hip/hip_math_constants.h>
#include <glm/glm.hpp>

#include "math/Utils.h"

using mk::core::complex;

__device__ complex makeComplex(float x, float y)
{
  complex c;

  c.x = x;
  c.y = y;

  return c;
}

__device__ complex add(complex a, complex b)
{
  return makeComplex(a.x + b.x, a.y + b.y);
}

__device__ complex mult(complex ab, complex cd)
{
    return makeComplex(ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x);
}

__device__ complex conj(complex arg)
{
    return makeComplex(arg.x, -arg.y);
}

__device__ complex eulerExp(float arg)
{
    return makeComplex(cosf(arg), sinf(arg));
}

__global__ void calculateSpectrumKernel(int sizeX, int sizeZ, float lengthX, float lengthZ, float gravity, float t, const complex* h0, complex* spectrum, complex* dispX, complex* dispZ, complex* gradX, complex* gradZ)
{
  int indexX = blockIdx.x * blockDim.x + threadIdx.x;
  int indexZ = blockIdx.y * blockDim.y + threadIdx.y;

  int indexXMirrored = sizeX - indexX - 1;
  int indexZMirrored = sizeZ - indexZ - 1;

  int index = indexZ * sizeX + indexX;
  int indexMirrored = indexZMirrored * sizeX + indexXMirrored;

  float kx = (indexX - sizeX / 2.0f) * (2.0f * mk::math::kPi / lengthX);
  float kz = (indexZ - sizeZ / 2.0f) * (2.0f * mk::math::kPi / lengthZ);
  float kLength = sqrt(kx * kx + kz * kz);
  float w = sqrt(gravity * kLength);

  spectrum[index] = add(mult(h0[index], eulerExp(w * t)),
                        mult(conj(h0[indexMirrored]), eulerExp(-w * t)));

  gradX[index] = makeComplex(-kx * spectrum[index].y, kx * spectrum[index].x);
  gradZ[index] = makeComplex(-kz * spectrum[index].y, kz * spectrum[index].x);

  if (kLength > 1e-12f)
  {
    dispX[index] = makeComplex(kx / kLength * spectrum[index].y, -kx / kLength * spectrum[index].x);
    dispZ[index] = makeComplex(kz / kLength * spectrum[index].y, -kz / kLength * spectrum[index].x);

    //jxx[index] = makeComplex(kx * kx / kLength * spectrum[index].x, kx * kx / kLength * spectrum[index].y);
    //jzz[index] = makeComplex(kz * kz / kLength * spectrum[index].x, kz * kz / kLength * spectrum[index].y);
    //jxz[index] = makeComplex(kx * kz / kLength * spectrum[index].x, kx * kz / kLength * spectrum[index].y);
  }
  else
  {
    dispX[index] = makeComplex(0.0f, 0.0f);
    dispZ[index] = makeComplex(0.0f, 0.0f);

    //jxx[index] = makeComplex(0.0f, 0.0f);
    //jzz[index] = makeComplex(0.0f, 0.0f);
    //jxz[index] = makeComplex(0.0f, 0.0f);
  }
}

__global__ void updateMeshKernel(int sizeX, int sizeZ, float dispFactor, const complex* h, complex* dispX, complex* dispZ, mk::core::VertexPN* mesh)
{
  int indexX = blockIdx.x * blockDim.x + threadIdx.x;
  int indexZ = blockIdx.y * blockDim.y + threadIdx.y;

  int index = indexZ * sizeX + indexX;

  float sign = ((indexX + indexZ) & 0x01) ? -1.0f : 1.0f;

  mesh[index].mPos.x = (indexX - sizeX / 2.0f) + dispFactor * sign * dispX[index].x;
  mesh[index].mPos.y = sign * h[index].x;
  mesh[index].mPos.z = (indexZ - sizeZ / 2.0f) + dispFactor * sign * dispZ[index].x;
}

__global__ void updateNormalsKernel(int sizeX, int sizeZ, complex* gradX, complex* gradZ, mk::core::VertexPN* mesh)
{
  int indexX = blockIdx.x * blockDim.x + threadIdx.x;
  int indexZ = blockIdx.y * blockDim.y + threadIdx.y;

  int index = indexZ * sizeX + indexX;

  float sign = ((indexX + indexZ) & 0x01) ? 1.0f : -1.0f; // Reverse sign here, since we need a - anyway

  mesh[index].mNormal = glm::normalize(glm::vec3(sign * gradX[index].x, 1.0f, sign * gradZ[index].x));
}

namespace mk
{
  namespace physics
  {
    namespace cuda
    {
      namespace
      {
        const int kBlocksPerSide = 16;
      }

      void calculateSpectrum(int sizeX, int sizeZ, float lengthX, float lengthZ, float gravity, float t, const complex* h0, complex* spectrum, complex* dispX, complex* dispZ, complex* gradX, complex* gradZ)
      {
        assert((sizeX % kBlocksPerSide == 0));
        assert((sizeZ % kBlocksPerSide == 0));

        dim3 grid(kBlocksPerSide, kBlocksPerSide, 1);
        dim3 block(sizeX / kBlocksPerSide, sizeZ / kBlocksPerSide, 1);

        calculateSpectrumKernel<<<grid, block>>>(sizeX, sizeZ, lengthX, lengthZ, gravity, t, h0, spectrum, dispX, dispZ, gradX, gradZ);
      }

      void updateMesh(int sizeX, int sizeZ, float dispFactor, const complex* h, complex* dispX, complex* dispZ, core::VertexPN* mesh)
      {
        assert((sizeX % kBlocksPerSide == 0));
        assert((sizeZ % kBlocksPerSide == 0));

        dim3 grid(kBlocksPerSide, kBlocksPerSide, 1);
        dim3 block(sizeX / kBlocksPerSide, sizeX / kBlocksPerSide, 1);

        updateMeshKernel<<<grid, block>>>(sizeX, sizeZ, dispFactor, h, dispX, dispZ, mesh);
      }

      void updateNormals(int sizeX, int sizeZ, complex* gradX, complex* gradZ, core::VertexPN* mesh)
      {
        assert((sizeX % kBlocksPerSide == 0));
        assert((sizeZ % kBlocksPerSide == 0));

        dim3 grid(kBlocksPerSide, kBlocksPerSide, 1);
        dim3 block(sizeX / kBlocksPerSide, sizeZ / kBlocksPerSide, 1);

        updateNormalsKernel<<<grid, block>>>(sizeX, sizeZ, gradX, gradZ, mesh);
      }
    }
  }
}
